#include "hip/hip_runtime.h"
/*
 * kernels.cu
 *
 *  Created on: Jun 20, 2017
 *      Author: kbmod-usr
 * funcs that use RESULTS_PER_PIXEL are:
 * searchImages, searchFilterImages. These functions are called in
 * deviceSearch and deviceSearchFilter
 */

#ifndef KERNELS_CU_
#define KERNELS_CU_
#define GPU_LC_FILTER 1
#define MAX_NUM_IMAGES 140

#include "common.h"
//#include "PointSpreadFunc.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <float.h>

namespace kbmod {


/*
 * Device kernel that convolves the provided image with the psf
 */
__global__ void convolvePSF(int width, int height,
    float *sourceImage, float *resultImage, float *psf,
    int psfRad, int psfDim, float psfSum, float maskFlag)
{
    // Find bounds of convolution area
    const int x = blockIdx.x*CONV_THREAD_DIM+threadIdx.x;
    const int y = blockIdx.y*CONV_THREAD_DIM+threadIdx.y;
    if (x < 0 || x > width-1 || y < 0 || y > height-1) return;
    const int minX = max(x-psfRad, 0);
    const int minY = max(y-psfRad, 0);
    const int maxX = min(x+psfRad, width-1);
    const int maxY = min(y+psfRad, height-1);

    // Read kernel
    float sum = 0.0;
    float psfPortion = 0.0;
    float center = sourceImage[y*width+x];
    if (center != NO_DATA) {
        for (int j=minY; j<=maxY; j++)
        {
            // #pragma unroll
            for (int i=minX; i<=maxX; i++)
            {
                float currentPixel = sourceImage[j*width+i];
                if (currentPixel != NO_DATA) {
                    float currentPSF = psf[(j-minY)*psfDim+i-minX];
                    psfPortion += currentPSF;
                    sum += currentPixel * currentPSF;
                }
            }
        }

        resultImage[y*width+x] = (sum*psfSum)/psfPortion;
    } else {
        // Leave masked pixel alone (these could be replaced here with zero)
        resultImage[y*width+x] = NO_DATA; // 0.0
    }
}

extern "C" void deviceConvolve(float *sourceImg, float *resultImg,
    int width, int height, float *psfKernel,
    int psfSize, int psfDim, int psfRadius, float psfSum)
{
    // Pointers to device memory //
    float *deviceKernel;
    float *deviceSourceImg;
    float *deviceResultImg;

    long pixelsPerImage = width*height;
    dim3 blocks(width/CONV_THREAD_DIM+1,height/CONV_THREAD_DIM+1);
    dim3 threads(CONV_THREAD_DIM,CONV_THREAD_DIM);

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceKernel, sizeof(float)*psfSize));
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float)*pixelsPerImage));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float)*pixelsPerImage));

    checkCudaErrors(hipMemcpy(deviceKernel, psfKernel,
        sizeof(float)*psfSize, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(deviceSourceImg, sourceImg,
        sizeof(float)*pixelsPerImage, hipMemcpyHostToDevice));

    convolvePSF<<<blocks, threads>>> (width, height, deviceSourceImg,
        deviceResultImg, deviceKernel, psfRadius, psfDim, psfSum, NO_DATA);

    checkCudaErrors(hipMemcpy(resultImg, deviceResultImg,
        sizeof(float)*pixelsPerImage, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceKernel));
    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

// Reads a single pixel from an image buffer
__device__ float readPixel(float* img, int x, int y, int width, int height)
{
    return (x<width && y<height) ? img[y*width+x] : NO_DATA;
}

__device__ float maxMasked(float pixel, float previousMax)
{
    return pixel == NO_DATA ? previousMax : max(pixel, previousMax);
}

__device__ float minMasked(float pixel, float previousMin)
{
    return pixel == NO_DATA ? previousMin : min(pixel, previousMin);
}

/*
 * Reduces the resolution of an image to 1/4 using max pooling
 */
__global__ void pool(int sourceWidth, int sourceHeight, float *source,
    int destWidth, int destHeight, float *dest, short mode)
{
    const int x = blockIdx.x*POOL_THREAD_DIM+threadIdx.x;
    const int y = blockIdx.y*POOL_THREAD_DIM+threadIdx.y;
    if (x>=destWidth || y>=destHeight) return;
    float mp;
    float pixel;
    if (mode == POOL_MAX) {
        mp = -FLT_MAX;
        pixel = readPixel(source, 2*x,   2*y,   sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y,   sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x,   2*y+1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y+1, sourceWidth, sourceHeight);
        mp = maxMasked(pixel, mp);
        if (mp == FLT_MIN) mp = NO_DATA;
    } else {
        mp = FLT_MAX;
        pixel = readPixel(source, 2*x,   2*y,   sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y,   sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x,   2*y+1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        pixel = readPixel(source, 2*x+1, 2*y+1, sourceWidth, sourceHeight);
        mp = minMasked(pixel, mp);
        if (mp == FLT_MAX) mp = NO_DATA;
    }

    dest[y*destWidth+x] = mp;
}

extern "C" void devicePool(int sourceWidth, int sourceHeight, float *source,
    int destWidth, int destHeight, float *dest, short mode)
{
    // Pointers to device memory //
    float *deviceSourceImg;
    float *deviceResultImg;

    dim3 blocks(destWidth/POOL_THREAD_DIM+1,destHeight/POOL_THREAD_DIM+1);
    dim3 threads(POOL_THREAD_DIM,POOL_THREAD_DIM);

    int srcPixCount = sourceWidth*sourceHeight;
    int destPixCount = destWidth*destHeight;

    // Allocate Device memory
    checkCudaErrors(hipMalloc((void **)&deviceSourceImg, sizeof(float)*srcPixCount));
    checkCudaErrors(hipMalloc((void **)&deviceResultImg, sizeof(float)*destPixCount));

    checkCudaErrors(hipMemcpy(deviceSourceImg, source,
        sizeof(float)*srcPixCount, hipMemcpyHostToDevice));

    pool<<<blocks, threads>>> (sourceWidth, sourceHeight, deviceSourceImg,
            destWidth, destHeight, deviceResultImg, mode);

    checkCudaErrors(hipMemcpy(dest, deviceResultImg,
        sizeof(float)*destPixCount, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceSourceImg));
    checkCudaErrors(hipFree(deviceResultImg));
}

/*
 * Searches through images (represented as a flat array of floats) looking for most likely
 * trajectories in the given list. Outputs a results image of best trajectories. Returns a
 * fixed number of results per pixel specified by RESULTS_PER_PIXEL
 */
__global__ void searchImages(int trajectoryCount, int width, int height,
    int imageCount, int minObservations, float *psiPhiImages,
    trajectory *trajectories, trajectory *results, float *imgTimes)
{

    // Get trajectory origin
    const unsigned short x = blockIdx.x*THREAD_DIM_X+threadIdx.x;
    const unsigned short y = blockIdx.y*THREAD_DIM_Y+threadIdx.y;
    trajectory best[RESULTS_PER_PIXEL];
    for (int r=0; r<RESULTS_PER_PIXEL; ++r)
    {
        best[r].lh = -1.0;
    }

    __shared__ float sImgTimes[512];
    int idx = threadIdx.x+threadIdx.y*THREAD_DIM_X;
    if (idx<imageCount) sImgTimes[idx] = imgTimes[idx];

    // Give up on any trajectories starting outside the image
    if (x >= width || y >= height)
    {
        return;
    }

    const unsigned int pixelsPerImage = width*height;

    // For each trajectory we'd like to search
    for (int t=0; t<trajectoryCount; ++t)
    {
        trajectory currentT;
        currentT.x = x;
        currentT.y = y;
        currentT.xVel = trajectories[t].xVel;
        currentT.yVel = trajectories[t].yVel;
        currentT.obsCount = 0;

        float psiSum = 0.0;
        float phiSum = 0.0;

        // Loop over each image and sample the appropriate pixel
        for (int i=0; i<imageCount; ++i)
        {
            float cTime = sImgTimes[i];
            int currentX = x + int(currentT.xVel*cTime+0.5);
            int currentY = y + int(currentT.yVel*cTime+0.5);
            // Test if trajectory goes out of image bounds
            // Branching could be avoided here by setting a
            // black image border and clamping coordinates
            if (currentX >= width || currentY >= height
                || currentX < 0 || currentY < 0)
            {
                // Penalize trajctories that leave edge
                //psiSum += -0.1;
                continue;
            }
            unsigned int pixel = (pixelsPerImage*i +
                 currentY*width +
                 currentX);

            //float cPsi = psiPhiImages[pixel];
            //float cPhi = psiPhiImages[pixel+1];
            float2 cPsiPhi = reinterpret_cast<float2*>(psiPhiImages)[pixel];
            if (cPsiPhi.x == NO_DATA) continue;

            currentT.obsCount++;
            psiSum += cPsiPhi.x;// < NO_DATA/2 /*== NO_DATA* / ? 0.0 : cPsiPhi.x;//min(cPsi,0.3);
            phiSum += cPsiPhi.y;// < NO_DATA/2 /*== NO_DATA* / ? 0.0 : cPsiPhi.y;
            //if (psiSum <= 0.0 && i>4) break;
        }
        // Just in case a phiSum is zero
        //phiSum += phiSum*1.0005+0.001;
        currentT.lh = psiSum/sqrt(phiSum);
        currentT.flux = /*2.0*fluxPix**/ psiSum/phiSum;
        trajectory temp;
        for (int r=0; r<RESULTS_PER_PIXEL; ++r)
        {
            if ( currentT.lh > best[r].lh &&
                 currentT.obsCount >= minObservations )
            {
                temp = best[r];
                best[r] = currentT;
                currentT = temp;
            }
        }
    }
    for (int r=0; r<RESULTS_PER_PIXEL; ++r)
    {
        results[ (y*width + x)*RESULTS_PER_PIXEL + r ] = best[r];
    }
    delete[] best;
}

extern "C" void
deviceSearch(int trajCount, int imageCount, int minObservations, int psiPhiSize,
    int resultsCount, trajectory *trajectoriesToSearch, trajectory *bestTrajects,
    float *imageTimes, float *interleavedPsiPhi, int width, int height)
{
    // Allocate Device memory
    trajectory *deviceTests;
    float *deviceImgTimes;
    float *devicePsiPhi;
    trajectory *deviceSearchResults;

    checkCudaErrors(hipMalloc((void **)&deviceTests, sizeof(trajectory)*trajCount));
    checkCudaErrors(hipMalloc((void **)&deviceImgTimes, sizeof(float)*imageCount));
    checkCudaErrors(hipMalloc((void **)&devicePsiPhi,
        sizeof(float)*psiPhiSize));
    checkCudaErrors(hipMalloc((void **)&deviceSearchResults,
        sizeof(trajectory)*resultsCount));

    // Copy trajectories to search
    checkCudaErrors(hipMemcpy(deviceTests, trajectoriesToSearch,
            sizeof(trajectory)*trajCount, hipMemcpyHostToDevice));

    // Copy image times
    checkCudaErrors(hipMemcpy(deviceImgTimes, imageTimes,
            sizeof(float)*imageCount, hipMemcpyHostToDevice));

    // Copy interleaved buffer of psi and phi images
    checkCudaErrors(hipMemcpy(devicePsiPhi, interleavedPsiPhi,
        sizeof(float)*psiPhiSize, hipMemcpyHostToDevice));

    //dim3 blocks(width,height);
    dim3 blocks(width/THREAD_DIM_X+1,height/THREAD_DIM_Y+1);
    dim3 threads(THREAD_DIM_X,THREAD_DIM_Y);


    // Launch Search
    searchImages<<<blocks, threads>>> (trajCount, width,
        height, imageCount, minObservations, devicePsiPhi,
        deviceTests, deviceSearchResults, deviceImgTimes);

    // Read back results
    checkCudaErrors(hipMemcpy(bestTrajects, deviceSearchResults,
                sizeof(trajectory)*resultsCount, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceTests));
    checkCudaErrors(hipFree(deviceImgTimes));
    checkCudaErrors(hipFree(deviceSearchResults));
    checkCudaErrors(hipFree(devicePsiPhi));
}

/*
 * Searches through images (represented as a flat array of floats) looking for most likely
 * trajectories in the given list. Outputs a results image of best trajectories. Returns a
 * fixed number of results per pixel specified by RESULTS_PER_PIXEL
 * filters results using a sigmaG-based filter and a central-moment filter.
 */
__global__ void searchFilterImages(int trajectoryCount, int width, int height,
        int imageCount, int minObservations, float *psiPhiImages,
        trajectory *trajectories, trajectory *results, float *imgTimes,
        float sGL0, float sGL1, float sigmaGCoeff, float minLH)
{

    // Get trajectory origin
    const unsigned short x = blockIdx.x*THREAD_DIM_X+threadIdx.x;
    const unsigned short y = blockIdx.y*THREAD_DIM_Y+threadIdx.y;
    float lcArray[MAX_NUM_IMAGES];
    float psiArray[MAX_NUM_IMAGES];
    float phiArray[MAX_NUM_IMAGES];
    int idxArray[MAX_NUM_IMAGES];
    float tmpSortValue;
    int tmpSortIdx;
    trajectory best[RESULTS_PER_PIXEL];
    for (int r=0; r<RESULTS_PER_PIXEL; ++r)
    {
        best[r].lh = -1.0;
    }

    __shared__ float sImgTimes[512];
    int idx = threadIdx.x+threadIdx.y*THREAD_DIM_X;
    if (idx<imageCount) sImgTimes[idx] = imgTimes[idx];

    // Give up on any trajectories starting outside the image
    if (x >= width || y >= height)
    {
        return;
    }

    const unsigned int pixelsPerImage = width*height;

    // For each trajectory we'd like to search
    for (int t=0; t<trajectoryCount; ++t)
    {
        trajectory currentT;
        currentT.x = x;
        currentT.y = y;
        currentT.xVel = trajectories[t].xVel;
        currentT.yVel = trajectories[t].yVel;
        currentT.obsCount = 0;

        float psiSum = 0.0;
        float phiSum = 0.0;

        // Loop over each image and sample the appropriate pixel
        for (int i=0; i<imageCount; ++i)
        {
            lcArray[i] = 0;
            psiArray[i] = 0;
            phiArray[i] = 0;
            idxArray[i] = i;
            float cTime = sImgTimes[i];
            int currentX = x + int(currentT.xVel*cTime+0.5);
            int currentY = y + int(currentT.yVel*cTime+0.5);
            // Test if trajectory goes out of image bounds
            // Branching could be avoided here by setting a
            // black image border and clamping coordinates
            if (currentX >= width || currentY >= height
                || currentX < 0 || currentY < 0)
            {
                // Penalize trajctories that leave edge
                //psiSum += -0.1;
                continue;
            }
            unsigned int pixel = (pixelsPerImage*i +
                 currentY*width +
                 currentX);

            //float cPsi = psiPhiImages[pixel];
            //float cPhi = psiPhiImages[pixel+1];
            float2 cPsiPhi = reinterpret_cast<float2*>(psiPhiImages)[pixel];
            if (cPsiPhi.x == NO_DATA) continue;

            currentT.obsCount++;
            psiSum += cPsiPhi.x;// < NO_DATA/2 /*== NO_DATA* / ? 0.0 : cPsiPhi.x;//min(cPsi,0.3);
            phiSum += cPsiPhi.y;// < NO_DATA/2 /*== NO_DATA* / ? 0.0 : cPsiPhi.y;
            psiArray[i] = cPsiPhi.x;
            phiArray[i] = cPsiPhi.y;
            if (cPsiPhi.y == 0.0)
            {
                lcArray[i] = 0;
            } else {
                lcArray[i] = cPsiPhi.x/cPsiPhi.y;
            }

            //if (psiSum <= 0.0 && i>4) break;
        }
        // Just in case a phiSum is zero
        //phiSum += phiSum*1.0005+0.001;
        currentT.lh = psiSum/sqrt(phiSum);
        currentT.flux = /*2.0*fluxPix**/ psiSum/phiSum;
        // Sort the the indexes (idxArray) of lcArray
        if (currentT.lh > minLH)
        {
            for (int j = 0; j < imageCount; j++)
            {
                for (int k = j+1; k < imageCount; k++)
                {
                     if (lcArray[idxArray[j]] > lcArray[idxArray[k]])
                     {
                         tmpSortIdx = idxArray[j];
                         idxArray[j] = idxArray[k];
                         idxArray[k] = tmpSortIdx;
/*
                         tmpSortValue = lcArray[j];
                         lcArray[j] = lcArray[k];
                         lcArray[k] = tmpSortValue;

                         tmpSortValue = psiArray[j];
                         psiArray[j] = psiArray[k];
                         psiArray[k] = tmpSortValue;

                         tmpSortValue = phiArray[j];
                         phiArray[j] = phiArray[k];
                         phiArray[k] = tmpSortValue;
                         */
                     }
                }
            }
            // 25th, 50th (median), and 75 percentiles
            int minKeepIndex = 0;
            int maxKeepIndex = imageCount - 1;
            int imgCountP1 = imageCount + 1;
            const int percentiles[3] = {
                int(imgCountP1 * sGL0 + 0.5) - 1,
                int(imgCountP1 * 0.5 + 0.5) - 1,
                int(imgCountP1 * sGL1 + 0.5) - 1};
            // 0.7413 comes from the inverse of the error function
            float sigmaG = sigmaGCoeff * (lcArray[idxArray[percentiles[2]]]
                    - lcArray[idxArray[percentiles[0]]]);
            float minValue = lcArray[idxArray[percentiles[1]]] - 2 * sigmaG;
            float maxValue = lcArray[idxArray[percentiles[1]]] + 2 * sigmaG;
            for (int i = 0; i <= percentiles[1]; i++)
            {
                int idx = idxArray[i];
                if (lcArray[idx] >= minValue)
                {
                    minKeepIndex = i;
                    break;
                }
            }
            for (int i = percentiles[1]+1; i<imageCount; i++)
            {
                int idx = idxArray[i];
                if (lcArray[idx] <= maxValue)
                {
                    maxKeepIndex = i;
                } else {
                    break;
                }
            }
            float newPsiSum = 0.0;
            float newPhiSum = 0.0;
            for (int i = minKeepIndex; i < maxKeepIndex+1; i++)
            {
                int idx = idxArray[i];
                newPsiSum += psiArray[idx];
                newPhiSum += phiArray[idx];
            }
            currentT.lh = newPsiSum/sqrt(newPhiSum);
            currentT.flux = newPsiSum/newPhiSum;
        }
        trajectory temp;
        for (int r=0; r<RESULTS_PER_PIXEL; ++r)
        {
            if ( currentT.lh > best[r].lh &&
                 currentT.obsCount >= minObservations )
            {
                temp = best[r];
                best[r] = currentT;
                currentT = temp;
            }
        }
    }
    for (int r=0; r<RESULTS_PER_PIXEL; ++r)
    {
        results[ (y*width + x)*RESULTS_PER_PIXEL + r ] = best[r];
    }
    delete[] best;
}

extern "C" void
deviceSearchFilter(
        int trajCount, int imageCount, int minObservations, int psiPhiSize,
        int resultsCount, trajectory *trajectoriesToSearch, trajectory *bestTrajects,
        float *imageTimes, float *interleavedPsiPhi, int width, int height,
        float sigmaGLims[2], float sigmaGCoeff, float centralMomLims[5],
        float minLH)
{
    // Allocate Device memory
    trajectory *deviceTests;
    float *deviceImgTimes;
    float *devicePsiPhi;
    // Allocate arrays for trajectory values. Needed for light curve filtering.
    /*
    float *devicePsiArray;
    float *devicePhiArray;
    float *deviceLCArray;
    checkCudaErrors(hipMalloc(&devicePsiArray,sizeof(float)*imageCount*trajCount));
    checkCudaErrors(hipMalloc(&devicePhiArray,sizeof(float)*imageCount*trajCount));
    checkCudaErrors(hipMalloc(&deviceLCArray,sizeof(float)*imageCount*trajCount));
    */
    trajectory *deviceSearchResults;

    checkCudaErrors(hipMalloc((void **)&deviceTests, sizeof(trajectory)*trajCount));
    checkCudaErrors(hipMalloc((void **)&deviceImgTimes, sizeof(float)*imageCount));
    checkCudaErrors(hipMalloc((void **)&devicePsiPhi,
        sizeof(float)*psiPhiSize));
    checkCudaErrors(hipMalloc((void **)&deviceSearchResults,
        sizeof(trajectory)*resultsCount));

    // Copy trajectories to search
    checkCudaErrors(hipMemcpy(deviceTests, trajectoriesToSearch,
            sizeof(trajectory)*trajCount, hipMemcpyHostToDevice));

    // Copy image times
    checkCudaErrors(hipMemcpy(deviceImgTimes, imageTimes,
            sizeof(float)*imageCount, hipMemcpyHostToDevice));

    // Copy interleaved buffer of psi and phi images
    checkCudaErrors(hipMemcpy(devicePsiPhi, interleavedPsiPhi,
        sizeof(float)*psiPhiSize, hipMemcpyHostToDevice));

    //dim3 blocks(width,height);
    dim3 blocks(width/THREAD_DIM_X+1,height/THREAD_DIM_Y+1);
    dim3 threads(THREAD_DIM_X,THREAD_DIM_Y);


    // Launch Search
    searchFilterImages<<<blocks, threads>>> (trajCount, width,
        height, imageCount, minObservations, devicePsiPhi,
        deviceTests, deviceSearchResults, deviceImgTimes, sigmaGLims[0],
        sigmaGLims[1], sigmaGCoeff, minLH);

    // Read back results
    checkCudaErrors(hipMemcpy(bestTrajects, deviceSearchResults,
                sizeof(trajectory)*resultsCount, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(deviceTests));
    checkCudaErrors(hipFree(deviceImgTimes));
    checkCudaErrors(hipFree(deviceSearchResults));
    checkCudaErrors(hipFree(devicePsiPhi));
}
extern "C" void
devicePooledSetup(int imageCount, int depth, long totalPixels, float *times, int *dimensions, float *interleavedImages,
        float **deviceTimes, float **deviceImages, int **deviceDimensions)
{
    checkCudaErrors(hipMalloc((void **)deviceTimes, sizeof(float)*imageCount));
    checkCudaErrors(hipMalloc((void **)deviceImages, sizeof(float)*totalPixels));
    checkCudaErrors(hipMalloc((void **)deviceDimensions, sizeof(int)*imageCount*2));

    // Copy image times
    checkCudaErrors(hipMemcpy(*deviceTimes, times,
            sizeof(float)*imageCount, hipMemcpyHostToDevice));

    // Copy interleaved buffer of pooled psi and phi images
    checkCudaErrors(hipMemcpy(*deviceImages, interleavedImages,
            sizeof(float)*totalPixels, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(*deviceDimensions, dimensions,
        sizeof(int)*imageCount*2, hipMemcpyHostToDevice));
}

extern "C" void
devicePooledTeardown(float **deviceTimes, float **deviceImages, int **dimensions)
{
    checkCudaErrors(hipFree(*deviceTimes));
    checkCudaErrors(hipFree(*deviceImages));
    checkCudaErrors(hipFree(*dimensions));
}

extern "C" void
deviceLHBatch(int imageCount, int depth, int regionCount, trajRegion *regions,
        float **deviceTimes, float **deviceImages, float **deviceDimensions)
{

}

} /* namespace kbmod */

#endif /* KERNELS_CU_ */
